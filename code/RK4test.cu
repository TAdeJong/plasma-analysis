#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

texture <float4, hipTextureType3D, hipReadModeElementType> dataTex;

const int N = 256;

const float spacing = 0.0245436930189;
const float origin = -3.12932085991;


//Definitions of vectortype operators
__device__ float3 operator+(const float3 &a, const float3 &b) {
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__device__ float3 operator*(const float &a, const float3 &b) {
	return make_float3(a*b.x, a*b.y, a*b.z);
}

__device__ float4 operator+(const float4 &a, const float4 &b) {
	return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

__device__ float4 operator*(const float &a, const float4 &b) {
	return make_float4(a*b.x, a*b.y, a*b.z,a*b.w);
}

inline __device__ float3 make_float3(float4 a) {
	return make_float3(a.x,a.y,a.z);
}

inline __device__ float4 tex3D(texture<float4, 3, hipReadModeElementType> tex, float3 a) {
	return tex3D(tex, a.x,a.y,a.z);
}

inline __device__ __host__ float3 Smiet2Tex(float4 locSmiet) {
	return make_float3((locSmiet.x-origin)/spacing+0.5,(locSmiet.y-origin)/spacing+0.5,(locSmiet.z-origin)/spacing+0.5);
}

//Do 1 RK4 step. Return een waarde in Smietcoords, input in Smietcoords
__device__ float4 RK4step(float4 loc, double dt ) {
	float3 loc3dTex = Smiet2Tex(loc);
	float4 k1 = tex3D(dataTex, loc3dTex);
	float4 k2 = tex3D(dataTex, loc3dTex+(dt*0.5/spacing)*make_float3(k1));
	float4 k3 = tex3D(dataTex, loc3dTex+(dt*0.5/spacing)*make_float3(k2));
	float4 k4 = tex3D(dataTex, loc3dTex+(dt/spacing)*make_float3(k3));
	return dt/6.0*(k1 + 2.0*(k2 + k3) + k4);
}

__global__ void RK4line(float4* lineoutput, double dt, unsigned int steps, float4 loc) {
	lineoutput[0] = tex3D(dataTex, Smiet2Tex(loc));
	for (unsigned int i=1; i < steps; i++) {
		loc = loc + RK4step(loc,dt);
		lineoutput[i] = loc;
	}
	return;
}

__global__ void readline(float4* lineoutput, unsigned int steps, float4 loc) {
	float3 loc3d = make_float3(loc);
	for (unsigned int i=0; i < steps; i++) {
		lineoutput[i] = tex3D(dataTex, Smiet2Tex(loc));
		loc.x+=spacing;
	}
	return;
}


void datagen (float4*** data) {
	//data[z][y][x]
	for (int i=0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) {
				(data[i][j][k]).x = - (origin + spacing*j);
				(data[i][j][k]).y = (origin + spacing*k);
				(data[i][j][k]).z = 0;
				(data[i][j][k]).w = 0;
			}
		}
	}
}

int main(void) {
	hipArray* dataArray;
//	std::cout << 1 << std::endl;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
	hipExtent extent = make_hipExtent(N , N, N);
	checkCudaErrors(hipMalloc3DArray(&dataArray, &channelDesc,extent));

	dataTex.filterMode = hipFilterModeLinear;
	//Generate data
	float4*** hostvfield; 
//	hostvfield = (float4*) malloc(N*N*N*sizeof(float4));
	hostvfield = (float4***) malloc(N*sizeof(float4**));
	hostvfield[0] = (float4**) malloc(N*N*sizeof(float4*));
	hostvfield[0][0] = (float4*) malloc(N*N*N*sizeof(float4));
//	std::cout << 2 << std::endl;
	for (int i=1; i < N; i++) {
		hostvfield[i] = (hostvfield[0] + i*N);
	}
	for (int i=0; i < N; i++) {
		for (int j=0; j < N; j++) {
			hostvfield[i][j] = (hostvfield[0][0] + (i*N*N + j*N));
		}
	}
//	std::cout << 2 << std::endl;
	datagen(hostvfield);
//	std::cout << extent.width << extent.height << extent.depth << std::endl;
//copy to device
	hipMemcpy3DParms copyParms = {0};
	copyParms.srcPtr = make_hipPitchedPtr((void *)hostvfield[0][0], extent.width* sizeof(float4), extent.height, extent.depth);
//	std::cout << 4 << std::endl;
	copyParms.dstArray = dataArray;
	copyParms.extent = extent;
	copyParms.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParms));
//	std::cout << 5 << std::endl;
	checkCudaErrors(hipBindTextureToArray(dataTex, dataArray, channelDesc));
	float4 *d_lines, *h_lines;
	double time = 3.141592653*2.0;
	int steps = 1000;
	int cores = 1;
	int blocks = 1;
	float dt = time/N;
	float4 startloc = {1,0,0,0};
	float3 locSmiet = Smiet2Tex(startloc);
	std::cout << "Starting point in texture coordinates: x=" << locSmiet.x << ", y=" << locSmiet.y << ", z=" << locSmiet.z << std::endl;
//	std::cout << hostvfield[127][127][127].x << std::endl;
	checkCudaErrors(hipMalloc(&d_lines,blocks*cores*steps*sizeof(float4)));

	h_lines = (float4*) malloc(blocks*cores*steps*sizeof(float4));
	RK4line<<<cores,blocks>>>(d_lines, dt, steps, startloc);
	checkCudaErrors(hipMemcpy(h_lines, d_lines, blocks*cores*steps*sizeof(float4), hipMemcpyDeviceToHost));
	for(unsigned int i=0; i<3; i++) {
		std::cout << "x= " << h_lines[i].x << "; y= "<< h_lines[i].y << " "<< h_lines[i].x*h_lines[i].x+h_lines[i].y*h_lines[i].y << std::endl;
	}
	free(hostvfield[0][0]);
	free(hostvfield[0]);
	free(hostvfield);
	
	return 0;
}

