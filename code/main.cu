#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include "constants.h"
#include "coordfunctions.h"
#include "integration.h"

//'Global' texture, declared as an external texture in integration.cu. Stores data on the device.
texture <float4, hipTextureType3D, hipReadModeElementType> dataTex;

/*	Generates a circular vectorfield around the origin for testing purposes.
	Note the order of the indices - the first index corresponds to the z coordinate,
	the middle to y and the last to x.
*/
void datagen (float4*** data) {
	//data[z][y][x]
	for (int i=0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) {
				(data[i][j][k]).x = - (origin + spacing*j);
				(data[i][j][k]).y = (origin + spacing*k);
				(data[i][j][k]).z = 0;
				(data[i][j][k]).w = 0;
			}
		}
	}
}

int main(void) {
	//Allocate array on device
	hipArray* dataArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
	hipExtent extent = make_hipExtent(N, N, N);
	checkCudaErrors(hipMalloc3DArray(&dataArray, &channelDesc,extent));

	//Set linear interpolation mode
	dataTex.filterMode = hipFilterModeLinear;

	//Allocate data array on host
	float4*** hostvfield; 
	hostvfield = (float4***) malloc(N*sizeof(float4**));
	hostvfield[0] = (float4**) malloc(N*N*sizeof(float4*));
	hostvfield[0][0] = (float4*) malloc(N*N*N*sizeof(float4));
	for (int i=1; i < N; i++) {
		hostvfield[i] = (hostvfield[0] + i*N);
	}
	for (int i=0; i < N; i++) {
		for (int j=0; j < N; j++) {
			hostvfield[i][j] = (hostvfield[0][0] + (i*N*N + j*N));
		}
	}

	//Generate data on host (used for testing)
	datagen(hostvfield);

	//Copy data to device
	hipMemcpy3DParms copyParms = {0};
	copyParms.srcPtr = make_hipPitchedPtr((void *)hostvfield[0][0], extent.width* sizeof(float4), extent.height, extent.depth);
	copyParms.dstArray = dataArray;
	copyParms.extent = extent;
	copyParms.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParms));

	//Copy our texture properties (linear interpolation, texture access) to data array on device
	checkCudaErrors(hipBindTextureToArray(dataTex, dataArray, channelDesc));

	//Declare pointers to arrays with line data (output of integration), one each on device and host
	float4 *d_lines, *h_lines;

	//Set integration parameters (end time, number of steps, etc.)
	double time = 3.141592653*6.0;
	int steps = 100000;
	float dt = time/steps;

	dim3 gridsize(1,1);
	dim3 blocksize(1,1);
	int blocks = gridsize.x * gridsize.y;
	int cores = blocksize.x * blocksize.y;
	float4 startloc = {0,0,0,0};
	float4 xvec = {1,0,0,0};
	float4 yvec = {0,1,0,0};

	//Allocate space on device to store integration output
	checkCudaErrors(hipMalloc(&d_lines,blocks*cores*steps*sizeof(float4)));

	//Allocate space on host to store integration output
	h_lines = (float4*) malloc(blocks*cores*steps*sizeof(float4));

	//Integrate the vector field
	RK4line<<<cores,blocks>>>(d_lines, dt, steps, startloc, xvec, yvec, gridsize);

	//Copy data from device to host
	checkCudaErrors(hipMemcpy(h_lines, d_lines, blocks*cores*steps*sizeof(float4), hipMemcpyDeviceToHost));

	//Print 100 samples from the line
	for(unsigned int i=0; i<100; i++) {
		std::cout << "x= " << h_lines[i*steps/100].x << "; y= "<< h_lines[i*steps/100].y << " "<< h_lines[i*steps/100].x*h_lines[i*steps/100].x+h_lines[i*steps/100].y*h_lines[i*steps/100].y << std::endl;
	}

	//Free host pointers
	free(hostvfield[0][0]);
	free(hostvfield[0]);
	free(hostvfield);
	
	return 0;
}

