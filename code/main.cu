#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "constants.h"
#include "coordfunctions.h"
#include "integration.h"

//'Global' texture, declared as an external texture in integration.cu. Stores data on the device.
texture <float4, hipTextureType3D, hipReadModeElementType> dataTex;

/*	Generates a circular vectorfield around the origin for testing purposes.
	Note the order of the indices - the first index corresponds to the z coordinate,
	the middle to y and the last to x.
*/
void datagen (float4*** data) {
	//data[z][y][x]
	for (int i=0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) {
				(data[i][j][k]).x = - (origin + spacing*j);
				(data[i][j][k]).y = (origin + spacing*k);
				(data[i][j][k]).z = 0;
				(data[i][j][k]).w = 0;
			}
		}
	}
}

int dataread (float4* data, const char* filename, float4 &origin){
	unsigned int i, j, k, datasize;
	unsigned int n_x =0, n_y=0, n_z=0;
	char kind[20];
	char name[20];
	char type[20];
	char rstr[80];
	float4 rspacing;
	FILE *dfp;

	dfp= fopen(filename, "r");
	for(unsigned int i=0; i<4; ++i) {
		fgets(rstr, 80, dfp);
		std::cout << rstr;
	}
	fscanf(dfp, "%s %u %u %u", rstr, &n_x, &n_y, &n_z);
	if(!( n_x == 256 && n_y == 256 && n_z ==256)) {
		std::cout<<"Warning: incorrect " << rstr << " read: expected 256, got: " << n_z << std::endl;
	}
	fscanf(dfp, "%s %f %f %f", rstr, &origin.x, &origin.y, &origin.z);
	fscanf(dfp, "%s %f %f %f", rstr, &rspacing.x, &rspacing.y, &rspacing.z);
	if(! (rspacing.x == rspacing.y && rspacing.y == rspacing.z)) {
		std::cout << "Warning: (unsupported) anisotrope spacing read!" << std::endl;
	}
	fscanf(dfp, "%s %u", rstr, &datasize);
	if(datasize != n_x*n_y*n_z) {
		std::cout<<"Error: " << rstr << "is not equal to n_x*n_y*n_z" << std::endl;
		return 1;
	}
	fscanf(dfp, "%s %s %s", kind, name, type);
	if(kind != "VECTORS" || name != "bfield" || type != "float") {
		std::cout << "Error: Incorrect kind, name or type" << std::endl;
		return 1;
	}
	for(unsigned int i=0; i<datasize; ++i) {
		float datapoint[3] = {0,0,0};
		fread(datapoint, sizeof(float), 3, dfp);
		data[i] = make_float4(datapoint[0],datapoint[1],datapoint[2],0);
	}
	std::cout << "Data read in was succesfull!" << std::endl;
	return 0;
}


void datawrite (const char* location, int steps, float4* h_lines){ 
    //write the first streamline to a file. Remember this is 32 bits when reading!
    FILE *fp;
    fp = fopen(location, "w");
    for (unsigned int i = 0; i<steps; i++){   //write only the first streamline
        fwrite(&h_lines[i], sizeof(float4), 1, fp);
    }
    fclose(fp);
    std::cout<<"streamline written!"<<std::endl;
}

int main(void) {
	//Allocate array on device
	hipArray* dataArray;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32,hipChannelFormatKindFloat);
	hipExtent extent = make_hipExtent(N, N, N);
	checkCudaErrors(hipMalloc3DArray(&dataArray, &channelDesc,extent));

	//Set linear interpolation mode
	dataTex.filterMode = hipFilterModeLinear;

	//Allocate data array on host
	float4*** hostvfield; 
	hostvfield = (float4***) malloc(N*sizeof(float4**));
	hostvfield[0] = (float4**) malloc(N*N*sizeof(float4*));
	hostvfield[0][0] = (float4*) malloc(N*N*N*sizeof(float4));
	for (int i=1; i < N; i++) {
		hostvfield[i] = (hostvfield[0] + i*N);
	}
	for (int i=0; i < N; i++) {
		for (int j=0; j < N; j++) {
			hostvfield[i][j] = (hostvfield[0][0] + (i*N*N + j*N));
		}
	}

	//Generate data on host (used for testing)
	datagen(hostvfield);

	//Copy data to device
	hipMemcpy3DParms copyParms = {0};
	copyParms.srcPtr = make_hipPitchedPtr((void *)hostvfield[0][0], extent.width* sizeof(float4), extent.height, extent.depth);
	copyParms.dstArray = dataArray;
	copyParms.extent = extent;
	copyParms.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParms));

	//Copy our texture properties (linear interpolation, texture access) to data array on device
	checkCudaErrors(hipBindTextureToArray(dataTex, dataArray, channelDesc));

	//Declare pointers to arrays with line data (output of integration), one each on device and host
	float4 *d_lines, *h_lines;

	//Set integration parameters (end time, number of steps, etc.)
	double time = 3.141592653*6.0;
	int steps = 100000;
	float dt = time/steps;

	dim3 gridsizeRK4(1,1);
	dim3 blocksizeRK4(8,8);
	int threadcountRK4 = gridsizeRK4.x*gridsizeRK4.y*blocksizeRK4.x*blocksizeRK4.y;
	float4 startloc = {1,0,0,0};
	float4 xvec = {1,0,0,0};
	float4 yvec = {0,1,0,0};

	//Allocate space on device to store integration output
	checkCudaErrors(hipMalloc(&d_lines, threadcountRK4*steps*sizeof(float4)));

	//Allocate space on host to store integration output
	h_lines = (float4*) malloc(threadcountRK4*steps*sizeof(float4));

	//Integrate the vector field
	RK4line<<<gridsizeRK4,blocksizeRK4>>>(d_lines, dt, steps, startloc, xvec, yvec, gridsizeRK4);

	//Copy data from device to host
	checkCudaErrors(hipMemcpy(h_lines, d_lines, threadcountRK4*steps*sizeof(float4), hipMemcpyDeviceToHost));

	//Print 100 samples from the line
	int index = 0;
	for(unsigned int i=0; i<100; i++) {
		index = 2*steps + i*steps/100;
		std::cout << "x= " << h_lines[index].x << "; y= "<< h_lines[index].y << " "<< h_lines[index].x*h_lines[index].x+h_lines[index].y*h_lines[index].y << std::endl;
	}
    
//    datawrite("../datadir/test.bin", steps, h_lines);
    dataread(hostvfield[0][0],"test.txt", startloc);
            
    //Free host pointers
	free(hostvfield[0][0]);
	free(hostvfield[0]);
	free(hostvfield);
        
	
	return 0;
}

