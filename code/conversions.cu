#include "hip/hip_runtime.h"
#include "coordfunctions.cuh"
#include "constants.cuh"
#include "conversions.cuh"
#include "hip/hip_vector_types.h"

extern texture <float4, hipTextureType3D, hipReadModeElementType> dataTex;

/*	Uses parallel computing to determine the origin (middle) of each of the field lines
	computed with RK4 and stored in lineoutput earlier.
	Warning: only works when the total number of threads used to call this function
	is a multiple of numberoflines, and their ratio is a divisor of the blocksize
	(so each RK4 line will be processed within a single block)
*/

__device__ int signdiff(float a, float b) {
	return (a < 0 && b >= 0) || (a>0 && b <=0);
}

//find the number of x=0 transitions in g_linedata, storing the result in g_sumdata. Only works for powers of 2 datasets and needs a minimum of sdata of 64*sizeof(float) (!)
__global__ void reducePC(float4* g_linedata, int* g_PCdata) {
	extern __shared__ int idata[];
	//load data from global data&texture to shared mem and perform cross product
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	idata[tid] = signdiff(g_linedata[i].x,g_linedata[i+1].x);
	__syncthreads();

	//do the reductions
	for( unsigned int s=blockDim.x/2; s>32; s>>=1) {//32 = warpsize
		if(tid < s) {
			idata[tid] += idata[tid+s];
		}

		__syncthreads();
	}
	if(tid<32) {// Warp's zijn SIMD gesynchroniseerd
		idata[tid] += idata[tid + 32];
		idata[tid] += idata[tid + 16];
		idata[tid] += idata[tid + 8];
		idata[tid] += idata[tid + 4];
		idata[tid] += idata[tid + 2];
		idata[tid] += idata[tid + 1];
	}
	//write result to global
	if(tid == 0) g_PCdata[blockIdx.x] = idata[0];
}

//Sum all elements in g_linedata, storing the result in g_sumdata. Only works for powers of 2 datasets
__global__ void reduceSum(float4* g_linedata, float4* g_sumdata) {
	extern __shared__ float4 sdata[];

	//load data from global data to shared mem and perform first reduction
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
	sdata[tid] = g_linedata[i]+g_linedata[i+blockDim.x];
	__syncthreads();

	//do the reductions
	unsigned int s=blockDim.x/2;
	for( ; s>32; s>>=1) {//32 = warpsize
		if(tid < s) {
			sdata[tid] += sdata[tid+s];
		}

		__syncthreads();
	}
	if(tid < s) {
		for( ; s>0; s>>=1) {// Warp's zijn SIMD gesynchroniseerd Loop-unroll would require a Template-use
			sdata[tid] += sdata[tid+s];
		}
	}

	//write result to global
	if(tid == 0) g_sumdata[blockIdx.x] = sdata[0];
}

/*	Sums all floats in g_linedata, storing the result in g_sumdata.
	Only works for powers of 2 datasets and needs a minimum of sdata of 64*sizeof(float4) (!)
	Identical to reduceSum for float4's, needless copying can be fixed with templates
	but requires clever inclusions of code throughout files. Maybe to be added later.
*/
__global__ void reduceSum(float* g_linedata, float* g_sumdata) {
	extern __shared__ float shdata[];

	//load data from global data to shared mem and perform first reduction
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
	shdata[tid] = g_linedata[i]+g_linedata[i+blockDim.x];
	__syncthreads();

	//do the reductions
	unsigned int s = blockDim.x/2;
	for( ; s>32; s>>=1) {//32 = warpsize
		if(tid < s) {
			shdata[tid] += shdata[tid+s];
		}

		__syncthreads();
	}
	if(tid < s) {
		for( ; s>0; s>>=1) {// Warp's zijn SIMD gesynchroniseerd Loop-unroll would require a Template-use
			shdata[tid] += shdata[tid+s];
		}
	}

	//write result to global
	if(tid == 0) g_sumdata[blockIdx.x] = shdata[0];
}


/*	Warning: absolutely useless!
	Mathematics is not correct, does not give normal to plane of torus!!
		DO NOT USE
	Give a third parameter to your kernellaunch for the size of sdata
__global__ void reduceNormal(float4* g_linedata, float4* g_normaldata) {//equivalent to doing the texture-fetch and cross product and applying reducesum
	extern __shared__ float4 sdata[];

	//load data from global data&texture to shared mem and perform cross product
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = make_float4(cross(make_float3(g_linedata[i]), make_float3(tex3D(dataTex, Smiet2Tex(g_linedata[i])))));
	__syncthreads();

	//do the reductions
	for( unsigned int s=blockDim.x/2; s>32; s>>=1) {//32 = warpsize
		if(tid < s) {
			sdata[tid] += sdata[tid+s];
		}

		__syncthreads();
	}
	if(tid<32) {// Warp's zijn SIMD gesynchroniseerd
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}
	//write result to global
	if(tid == 0) g_normaldata[blockIdx.x] = sdata[0];
}*/

__global__ void winding(float4* g_linedata, float4* g_windingdata, float4* origin, float* g_rdata, unsigned int steps) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int modifier = min(i%steps,1);
	float r_t = g_rdata[i/steps];
	float4 locCord = Cart2Tor(ShiftCoord(g_linedata[i], origin[i/steps]), r_t);
	locCord -= Cart2Tor(ShiftCoord(g_linedata[i-modifier], origin[i/steps]), r_t);
	//lelijk en langzaam, maar mijn bit-wise magic is niet genoeg om dit netjes te doen
	if(locCord.y > PI) {
		locCord.y -= 2*PI;
	} else if (locCord.y< -1*PI) {
		locCord.y += 2*PI;
	}
	if(locCord.z > PI) {
		locCord.z -= 2*PI;
	} else if (locCord.z < -1*PI) {
		locCord.z += 2*PI;
	}
	g_windingdata[i] =  locCord;
}

__global__ void divide(float* enumerator, float* denominator, float* output) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	output[i] = enumerator[i]/denominator[i];
}
