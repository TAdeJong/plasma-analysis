
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void) {
	int *d_a, *d_b, *d_c;
	size_t size = 2*sizeof(int);
	int a[2] = {1,2};
	int b[2] = {1,2};
	int c[2] = {0,0};
	int test;
	hipGetDeviceCount(&test);
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	printf("c[0]%d\n",c[0]);
	printf("c[1]%d\n",c[1]);
	add<<<1,2>>>(d_a, d_b, d_c);
	a[1] = 6;
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	hipMemcpy(&b, d_b, size, hipMemcpyDeviceToHost);
	hipMemcpy(&a, d_a, size, hipMemcpyDeviceToHost);
	printf("c[0]%d\n",c[0]);
	printf("c[1]%d\n",c[1]);
	printf("%p\n", &a);
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c); 
	return 0;
}
